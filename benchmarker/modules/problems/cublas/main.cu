//#include <stdio.h>
//#include <stdlib.h>
#include <iostream>
#include <cblas.h>
#include <omp.h>
#include <chrono> 
#include "config.h"
#include "args.hpp"
#include <unistd.h>

using namespace std::chrono; 


int main(int argc, char * argv[]) {
    size_t m, n, k;
    float * A, *B, *C;
    double dtime;
    args_to_matrices(argc, argv, m, n, k, A, B, C);
    auto start = high_resolution_clock::now(); 
    // move to gpu
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * n * sizeof(float));
    hipMalloc(&d_B, n * k * sizeof(float));
    hipMalloc(&d_C, m * k * sizeof(float));
    hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);
     
    // call cublas
    // sync
    // cblas_sgemm(CblasColMajor, CblasNoTrans, CblasTrans, m, k, n, 1, A, m, B, k, 1, C, m);
    sleep(1);
    auto stop = high_resolution_clock::now();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    std::chrono::duration<double> seconds = (stop - start); 
    dtime = seconds.count();
    double gflop = (2.0 * m * n * k) / (1024 * 1024 * 1024);
    double gflops = gflop / dtime;
    printf("gflops: \t%f\n", gflop);
    printf("time: \t%f\n", dtime);
    printf("ips: \t%f\n", 1 / dtime);
    printf("gflops/s: \t%f\n", gflops);
    return 0;
}

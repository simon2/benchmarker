#include <iostream>
#include <hipblas.h>
#include <chrono>
//#include <hip/hip_runtime.h> 
//#include <cuda_device_runtime_api.h> 
#include "config.h"
#include "args.hpp"
#include <unistd.h>

using namespace std::chrono; 


int main(int argc, char * argv[]) {
    size_t m, n, k;
    float * A, *B, *C;
    double dtime;
    args_to_matrices(argc, argv, m, n, k, A, B, C);
    auto start = high_resolution_clock::now(); 
    // move to gpu
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * n * sizeof(float));
    hipMalloc(&d_B, n * k * sizeof(float));
    hipMalloc(&d_C, m * k * sizeof(float));
    hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);
    // TODO: add error checking
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    int lda=m,ldb=k,ldc=m;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
    hipDeviceSynchronize();
    // cblas_sgemm(CblasColMajor, CblasNoTrans, CblasTrans, m, k, n, 1, A, m, B, k, 1, C, m);
    auto stop = high_resolution_clock::now();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    std::chrono::duration<double> seconds = (stop - start); 
    dtime = seconds.count();
    double gflop = (2.0 * m * n * k) / (1024 * 1024 * 1024);
    double gflops = gflop / dtime;
    printf("gflops: \t%f\n", gflop);
    printf("time: \t%f\n", dtime);
    printf("ips: \t%f\n", 1 / dtime);
    printf("gflops/s: \t%f\n", gflops);
    return 0;
}
